#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "my_jacobi_2d_kernel.hu"
//#include<stdio.h>
int A[4][4];
int B[4][4];
int main()
{	

	int t, i, j,tm,tj;
	  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

	    int *dev_A;
	    int *dev_B;
	    
	    cudaCheckReturn(hipMalloc((void **) &dev_A, (4) * (4) * sizeof(int)));
	    cudaCheckReturn(hipMalloc((void **) &dev_B, (4) * (4) * sizeof(int)));
	    
	    
	    
	    cudaCheckReturn(hipMemcpy(dev_A, A, (4) * (4) * sizeof(int), hipMemcpyHostToDevice));
	    cudaCheckReturn(hipMemcpy(dev_B, B, (4) * (4) * sizeof(int), hipMemcpyHostToDevice));
	    {
	      dim3 k0_dimBlock(3, 3);
	      dim3 k0_dimGrid(1, 1);
	      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B);
	      cudaCheckKernel();
	    }
	    
	    
	    {
	      dim3 k1_dimBlock(3, 3);
	      dim3 k1_dimGrid(1, 1);
	      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, dev_B);
	      cudaCheckKernel();
	    }
	    
	    
	    cudaCheckReturn(hipMemcpy(A, dev_A, (4) * (4) * sizeof(int), hipMemcpyDeviceToHost));
	    cudaCheckReturn(hipMemcpy(B, dev_B, (4) * (4) * sizeof(int), hipMemcpyDeviceToHost));
	    
	    
	    cudaCheckReturn(hipFree(dev_A));
	    cudaCheckReturn(hipFree(dev_B));
	  }

  return 0;
}
