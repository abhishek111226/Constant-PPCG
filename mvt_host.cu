#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "mvt_kernel.hu"
#define _PB_N 100
int x1[100];
int x2[100];
int y_1[100];
int y_2[100];
int A[100][100];
int main()
{
  int i,j;
  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    int *dev_A;
    int *dev_x1;
    int *dev_x2;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (100) * (100) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_x1, (100) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_x2, (100) * sizeof(int)));
    
    
    hipMemcpyToSymbol(HIP_SYMBOL(const_y_1), y_1, (100) * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(const_y_2), y_2, (100) * sizeof(int));
    
    cudaCheckReturn(hipMemcpy(dev_A, A, (100) * (100) * sizeof(int), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x1, x1, (100) * sizeof(int), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x2, x2, (100) * sizeof(int), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(32);
      dim3 k0_dimGrid(4);
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_x1);
      cudaCheckKernel();
    }
    
    
    {
      dim3 k1_dimBlock(32);
      dim3 k1_dimGrid(4);
      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, dev_x2);
      cudaCheckKernel();
    }
    
    
    cudaCheckReturn(hipMemcpy(x1, dev_x1, (100) * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(x2, dev_x2, (100) * sizeof(int), hipMemcpyDeviceToHost));
    
    
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_x1));
    cudaCheckReturn(hipFree(dev_x2));
  }
}

/*enum RWbar 
{
	write,	0
	read,	1
	invalid,2 
	error,  3 
	none,   4
	read_inside_loop 5
}; */
