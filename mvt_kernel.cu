#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "mvt_kernel.hu"
__global__ void kernel0(int *A, int *x1)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 99; c1 += 32) {
      if (32 * b0 + t0 <= 99)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 99); c3 += 1)
          x1[32 * b0 + t0] = (x1[32 * b0 + t0] + (A[(32 * b0 + t0) * 100 + (c1 + c3)] * const_y_1[c1 + c3]));
      __syncthreads();
    }
}
__global__ void kernel1(int *A, int *x2)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 99; c1 += 32) {
      if (32 * b0 + t0 <= 99)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 99); c3 += 1)
          x2[32 * b0 + t0] = (x2[32 * b0 + t0] + (A[(c1 + c3) * 100 + (32 * b0 + t0)] * const_y_2[c1 + c3]));
      __syncthreads();
    }
}
