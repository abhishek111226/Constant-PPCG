#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "chol_kernel.hu"
__global__ void kernel0(int *A, int c0)
{

    {
      for (int c1 = 0; c1 < c0; c1 += 1)
        A[c0 * 3 + c0] -= (A[c0 * 3 + c1] * A[c0 * 3 + c1]);
      if (c0 == 1)
        A[2 * 3 + 1] -= (A[2 * 3 + 0] * A[1 * 3 + 0]);
    }
}
__global__ void kernel1(int *A, int c0)
{

    A[c0 * 3 + c0] = A[c0 * 3 + c0];
}
__global__ void kernel2(int *A, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (t0 >= c0 + 1)
      A[t0 * 3 + c0] /= A[c0 * 3 + c0];
}
void init_const_scop_0()
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)




}

