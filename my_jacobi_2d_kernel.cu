#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "my_jacobi_2d_kernel.hu"
__global__ void kernel0(int *A, int *B)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    if (t0 >= 1 && t1 >= 1)
      B[t0 * 4 + t1] = ((((A[t0 * 4 + t1] + A[t0 * 4 + (t1 - 1)]) + A[t0 * 4 + (t1 + 1)]) + A[(t0 + 1) * 4 + t1]) + A[(t0 - 1) * 4 + t1]);
}
__global__ void kernel1(int *A, int *B)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    if (t0 >= 1 && t1 >= 1)
      A[t0 * 4 + t1] = ((((B[t0 * 4 + t1] + B[t0 * 4 + (t1 - 1)]) + B[t0 * 4 + (t1 + 1)]) + B[(t0 + 1) * 4 + t1]) + B[(t0 - 1) * 4 + t1]);
}
