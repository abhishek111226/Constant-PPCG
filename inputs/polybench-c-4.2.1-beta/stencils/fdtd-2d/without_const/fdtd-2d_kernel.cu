#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "fdtd-2d_kernel.hu"
__global__ void kernel0(double _fict_[40], double ey[60][80], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 <= 79)
      ey[0][32 * b0 + t0] = _fict_[c0];
}
__global__ void kernel1(double ey[60][80], double hz[60][80], int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 59)
      for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 79); c4 += 16)
        ey[32 * b0 + t0][32 * b1 + c4] = (ey[32 * b0 + t0][32 * b1 + c4] - (0.5 * (hz[32 * b0 + t0][32 * b1 + c4] - hz[32 * b0 + t0 - 1][32 * b1 + c4])));
}
__global__ void kernel2(double ex[60][80], double hz[60][80], int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    if (32 * b0 + t0 <= 59)
      for (int c4 = ppcg_max(t1, ((t1 + 15) % 16) - 32 * b1 + 1); c4 <= ppcg_min(31, -32 * b1 + 79); c4 += 16)
        ex[32 * b0 + t0][32 * b1 + c4] = (ex[32 * b0 + t0][32 * b1 + c4] - (0.5 * (hz[32 * b0 + t0][32 * b1 + c4] - hz[32 * b0 + t0][32 * b1 + c4 - 1])));
}
__global__ void kernel3(double ex[60][80], double ey[60][80], double hz[60][80], int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    if (32 * b0 + t0 <= 58)
      for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 78); c4 += 16)
        hz[32 * b0 + t0][32 * b1 + c4] = (hz[32 * b0 + t0][32 * b1 + c4] - (0.69999999999999996 * (((ex[32 * b0 + t0][32 * b1 + c4 + 1] - ex[32 * b0 + t0][32 * b1 + c4]) + ey[32 * b0 + t0 + 1][32 * b1 + c4]) - ey[32 * b0 + t0][32 * b1 + c4])));
}
