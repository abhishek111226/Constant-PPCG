#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "fdtd-2d_kernel.hu"
__global__ void kernel0(double ey[60][80], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 <= 79)
      ey[0][32 * b0 + t0] = const__fict_[c0];
}
__global__ void kernel1(double ey[60][80], double hz[60][80], int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 59)
      for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 79); c4 += 16)
        ey[32 * b0 + t0][32 * b1 + c4] = (ey[32 * b0 + t0][32 * b1 + c4] - (0.5 * (hz[32 * b0 + t0][32 * b1 + c4] - hz[32 * b0 + t0 - 1][32 * b1 + c4])));
}
__global__ void kernel2(double ex[60][80], double hz[60][80], int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    if (32 * b0 + t0 <= 59)
      for (int c4 = ppcg_max(t1, ((t1 + 15) % 16) - 32 * b1 + 1); c4 <= ppcg_min(31, -32 * b1 + 79); c4 += 16)
        ex[32 * b0 + t0][32 * b1 + c4] = (ex[32 * b0 + t0][32 * b1 + c4] - (0.5 * (hz[32 * b0 + t0][32 * b1 + c4] - hz[32 * b0 + t0][32 * b1 + c4 - 1])));
}
__global__ void kernel3(double ex[60][80], double ey[60][80], double hz[60][80], int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    if (32 * b0 + t0 <= 58)
      for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 78); c4 += 16)
        hz[32 * b0 + t0][32 * b1 + c4] = (hz[32 * b0 + t0][32 * b1 + c4] - (0.69999999999999996 * (((ex[32 * b0 + t0][32 * b1 + c4 + 1] - ex[32 * b0 + t0][32 * b1 + c4]) + ey[32 * b0 + t0 + 1][32 * b1 + c4]) - ey[32 * b0 + t0][32 * b1 + c4])));
}
void init_const_scop_0(double _fict_[40])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const__fict_), _fict_, (40) * sizeof(double)));


}

