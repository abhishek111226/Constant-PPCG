#include <stdio.h>
#define HOSTCODE true 
#include "fdtd-2d_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* fdtd-2d.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "fdtd-2d.h"


/* Array initialization. */
static
void init_array (int tmax,
		 int nx,
		 int ny,
		 DATA_TYPE POLYBENCH_2D(ex,NX,NY,nx,ny),
		 DATA_TYPE POLYBENCH_2D(ey,NX,NY,nx,ny),
		 DATA_TYPE POLYBENCH_2D(hz,NX,NY,nx,ny),
		 DATA_TYPE POLYBENCH_1D(_fict_,TMAX,tmax))
{
  int i, j;

  for (i = 0; i < tmax; i++)
    _fict_[i] = (DATA_TYPE) i;
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++)
      {
	ex[i][j] = ((DATA_TYPE) i*(j+1)) / nx;
	ey[i][j] = ((DATA_TYPE) i*(j+2)) / ny;
	hz[i][j] = ((DATA_TYPE) i*(j+3)) / nx;
      }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int nx,
		 int ny,
		 DATA_TYPE POLYBENCH_2D(ex,NX,NY,nx,ny),
		 DATA_TYPE POLYBENCH_2D(ey,NX,NY,nx,ny),
		 DATA_TYPE POLYBENCH_2D(hz,NX,NY,nx,ny))
{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("ex");
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++) {
      if ((i * nx + j) % 20 == 0) fprintf(POLYBENCH_DUMP_TARGET, "\n");
      fprintf(POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, ex[i][j]);
    }
  POLYBENCH_DUMP_END("ex");
  POLYBENCH_DUMP_FINISH;

  POLYBENCH_DUMP_BEGIN("ey");
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++) {
      if ((i * nx + j) % 20 == 0) fprintf(POLYBENCH_DUMP_TARGET, "\n");
      fprintf(POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, ey[i][j]);
    }
  POLYBENCH_DUMP_END("ey");

  POLYBENCH_DUMP_BEGIN("hz");
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++) {
      if ((i * nx + j) % 20 == 0) fprintf(POLYBENCH_DUMP_TARGET, "\n");
      fprintf(POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, hz[i][j]);
    }
  POLYBENCH_DUMP_END("hz");
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_fdtd_2d(int tmax,
		    int nx,
		    int ny,
		    DATA_TYPE POLYBENCH_2D(ex,NX,NY,nx,ny),
		    DATA_TYPE POLYBENCH_2D(ey,NX,NY,nx,ny),
		    DATA_TYPE POLYBENCH_2D(hz,NX,NY,nx,ny),
		    DATA_TYPE POLYBENCH_1D(_fict_,TMAX,tmax))
{
  int t, i, j;

  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    float *dev__fict_;
    float (*dev_ex)[80];
    float (*dev_ey)[80];
    float (*dev_hz)[80];
    
    cudaCheckReturn(hipMalloc((void **) &dev__fict_, (40) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_ex, (60) * (80) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_ey, (60) * (80) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_hz, (60) * (80) * sizeof(float)));
    
    
    cudaCheckReturn(hipMemcpy(dev__fict_, _fict_, (40) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_ex, ex, (60) * (80) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_ey, ey, (60) * (80) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_hz, hz, (60) * (80) * sizeof(float), hipMemcpyHostToDevice));
    for (int c0 = 0; c0 <= 39; c0 += 1) {
      {
        dim3 k0_dimBlock(32);
        dim3 k0_dimGrid(3);
        kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev__fict_, dev_ey, c0);
        cudaCheckKernel();
      }
      
      
      {
        dim3 k1_dimBlock(16, 32);
        dim3 k1_dimGrid(3, 2);
        kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_ey, dev_hz, c0);
        cudaCheckKernel();
      }
      
      
      {
        dim3 k2_dimBlock(16, 32);
        dim3 k2_dimGrid(3, 2);
        kernel2 <<<k2_dimGrid, k2_dimBlock>>> (dev_ex, dev_hz, c0);
        cudaCheckKernel();
      }
      
      
      {
        dim3 k3_dimBlock(16, 32);
        dim3 k3_dimGrid(3, 2);
        kernel3 <<<k3_dimGrid, k3_dimBlock>>> (dev_ex, dev_ey, dev_hz, c0);
        cudaCheckKernel();
      }
      
      
    }
    cudaCheckReturn(hipMemcpy(ex, dev_ex, (60) * (80) * sizeof(float), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(ey, dev_ey, (60) * (80) * sizeof(float), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(hz, dev_hz, (60) * (80) * sizeof(float), hipMemcpyDeviceToHost));
    
    
    cudaCheckReturn(hipFree(dev__fict_));
    cudaCheckReturn(hipFree(dev_ex));
    cudaCheckReturn(hipFree(dev_ey));
    cudaCheckReturn(hipFree(dev_hz));
  }
}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int tmax = TMAX;
  int nx = NX;
  int ny = NY;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(ex,DATA_TYPE,NX,NY,nx,ny);
  POLYBENCH_2D_ARRAY_DECL(ey,DATA_TYPE,NX,NY,nx,ny);
  POLYBENCH_2D_ARRAY_DECL(hz,DATA_TYPE,NX,NY,nx,ny);
  POLYBENCH_1D_ARRAY_DECL(_fict_,DATA_TYPE,TMAX,tmax);

  /* Initialize array(s). */
  init_array (tmax, nx, ny,
	      POLYBENCH_ARRAY(ex),
	      POLYBENCH_ARRAY(ey),
	      POLYBENCH_ARRAY(hz),
	      POLYBENCH_ARRAY(_fict_));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_fdtd_2d (tmax, nx, ny,
		  POLYBENCH_ARRAY(ex),
		  POLYBENCH_ARRAY(ey),
		  POLYBENCH_ARRAY(hz),
		  POLYBENCH_ARRAY(_fict_));


  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(nx, ny, POLYBENCH_ARRAY(ex),
				    POLYBENCH_ARRAY(ey),
				    POLYBENCH_ARRAY(hz)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(ex);
  POLYBENCH_FREE_ARRAY(ey);
  POLYBENCH_FREE_ARRAY(hz);
  POLYBENCH_FREE_ARRAY(_fict_);

  return 0;
}
