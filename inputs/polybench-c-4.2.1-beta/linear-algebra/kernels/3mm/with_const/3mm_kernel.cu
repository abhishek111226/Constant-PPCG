#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "3mm_kernel.hu"
__global__ void kernel0(double A[40][60], double E[40][50])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 59; c2 += 32) {
      if (32 * b0 + t0 <= 39)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 49); c4 += 16) {
          if (c2 == 0)
            E[32 * b0 + t0][32 * b1 + c4] = 0.;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 59); c5 += 1)
            E[32 * b0 + t0][32 * b1 + c4] += (A[32 * b0 + t0][c2 + c5] * const_B[c2 + c5][32 * b1 + c4]);
        }
      __syncthreads();
    }
}
__global__ void kernel1(double C[50][80], double F[50][70])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 79; c2 += 32) {
      if (32 * b0 + t0 <= 49)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 69); c4 += 16) {
          if (c2 == 0)
            F[32 * b0 + t0][32 * b1 + c4] = 0.;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 79); c5 += 1)
            F[32 * b0 + t0][32 * b1 + c4] += (C[32 * b0 + t0][c2 + c5] * const_D[c2 + c5][32 * b1 + c4]);
        }
      __syncthreads();
    }
}
__global__ void kernel2(double E[40][50], double F[50][70], double G[40][70])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 49; c2 += 32) {
      if (32 * b0 + t0 <= 39)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 69); c4 += 16) {
          if (c2 == 0)
            G[32 * b0 + t0][32 * b1 + c4] = 0.;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 49); c5 += 1)
            G[32 * b0 + t0][32 * b1 + c4] += (E[32 * b0 + t0][c2 + c5] * F[c2 + c5][32 * b1 + c4]);
        }
      __syncthreads();
    }
}
void init_const_scop_0(double B[60][50],double D[80][70])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_B), B, (60) * (50) * sizeof(double)));
cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_D), D, (80) * (70) * sizeof(double)));


}

