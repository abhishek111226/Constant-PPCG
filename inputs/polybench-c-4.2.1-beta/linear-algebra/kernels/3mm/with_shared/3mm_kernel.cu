#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "3mm_kernel.hu"
__global__ void kernel0(float A[40][60], float B[60][50], float E[40][50])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_A[32][32];
    __shared__ float shared_B[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 59; c2 += 32) {
      if (32 * b0 + t0 <= 39)
        for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 59); c4 += 16)
          shared_A[t0][c4] = A[32 * b0 + t0][c2 + c4];
      if (t0 + c2 <= 59)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 49); c4 += 16)
          shared_B[t0][c4] = B[t0 + c2][32 * b1 + c4];
      __syncthreads();
      if (32 * b0 + t0 <= 39)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 49); c4 += 16) {
          if (c2 == 0)
            E[32 * b0 + t0][32 * b1 + c4] = 0.F;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 59); c5 += 1)
            E[32 * b0 + t0][32 * b1 + c4] += (shared_A[t0][c5] * shared_B[c5][c4]);
        }
      __syncthreads();
    }
}
__global__ void kernel1(float C[50][80], float D[80][70], float F[50][70])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_C[32][32];
    __shared__ float shared_D[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 79; c2 += 32) {
      if (32 * b0 + t0 <= 49)
        for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 79); c4 += 16)
          shared_C[t0][c4] = C[32 * b0 + t0][c2 + c4];
      if (t0 + c2 <= 79)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 69); c4 += 16)
          shared_D[t0][c4] = D[t0 + c2][32 * b1 + c4];
      __syncthreads();
      if (32 * b0 + t0 <= 49)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 69); c4 += 16) {
          if (c2 == 0)
            F[32 * b0 + t0][32 * b1 + c4] = 0.F;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 79); c5 += 1)
            F[32 * b0 + t0][32 * b1 + c4] += (shared_C[t0][c5] * shared_D[c5][c4]);
        }
      __syncthreads();
    }
}
__global__ void kernel2(float E[40][50], float F[50][70], float G[40][70])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_E[32][32];
    __shared__ float shared_F[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 49; c2 += 32) {
      if (32 * b0 + t0 <= 39)
        for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 49); c4 += 16)
          shared_E[t0][c4] = E[32 * b0 + t0][c2 + c4];
      if (t0 + c2 <= 49)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 69); c4 += 16)
          shared_F[t0][c4] = F[t0 + c2][32 * b1 + c4];
      __syncthreads();
      if (32 * b0 + t0 <= 39)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 69); c4 += 16) {
          if (c2 == 0)
            G[32 * b0 + t0][32 * b1 + c4] = 0.F;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 49); c5 += 1)
            G[32 * b0 + t0][32 * b1 + c4] += (shared_E[t0][c5] * shared_F[c5][c4]);
        }
      __syncthreads();
    }
}
