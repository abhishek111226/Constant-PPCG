#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "mvt_kernel.hu"
__global__ void kernel0(double A[120][120], double x1[120], double y_1[120])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 119; c1 += 32) {
      if (32 * b0 + t0 <= 119)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 119); c3 += 1)
          x1[32 * b0 + t0] = (x1[32 * b0 + t0] + (A[32 * b0 + t0][c1 + c3] * y_1[c1 + c3]));
      __syncthreads();
    }
}
__global__ void kernel1(double A[120][120], double x2[120], double y_2[120])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 119; c1 += 32) {
      if (32 * b0 + t0 <= 119)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 119); c3 += 1)
          x2[32 * b0 + t0] = (x2[32 * b0 + t0] + (A[c1 + c3][32 * b0 + t0] * y_2[c1 + c3]));
      __syncthreads();
    }
}
