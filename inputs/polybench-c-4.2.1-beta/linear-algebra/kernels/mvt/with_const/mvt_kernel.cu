#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "mvt_kernel.hu"
__global__ void kernel0(double A[120][120], double x1[120])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 119; c1 += 32) {
      if (32 * b0 + t0 <= 119)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 119); c3 += 1)
          x1[32 * b0 + t0] = (x1[32 * b0 + t0] + (A[32 * b0 + t0][c1 + c3] * const_y_1[c1 + c3]));
      __syncthreads();
    }
}
__global__ void kernel1(double A[120][120], double x2[120])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 119; c1 += 32) {
      if (32 * b0 + t0 <= 119)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 119); c3 += 1)
          x2[32 * b0 + t0] = (x2[32 * b0 + t0] + (A[c1 + c3][32 * b0 + t0] * const_y_2[c1 + c3]));
      __syncthreads();
    }
}
void init_const_scop_0(double y_1[120],double y_2[120])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_y_1), y_1, (120) * sizeof(double)));
cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_y_2), y_2, (120) * sizeof(double)));


}

