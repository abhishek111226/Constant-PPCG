#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "bicg_kernel.hu"
__global__ void kernel0(double A[124][116], double s[116])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 123; c1 += 32) {
      if (32 * b0 + t0 <= 115 && c1 == 0)
        s[32 * b0 + t0] = 0;
      if (32 * b0 + t0 <= 115)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 123); c3 += 1)
          s[32 * b0 + t0] = (s[32 * b0 + t0] + (const_r[c1 + c3] * A[c1 + c3][32 * b0 + t0]));
      __syncthreads();
    }
}
__global__ void kernel1(double A[124][116], double q[124])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 115; c1 += 32) {
      if (32 * b0 + t0 <= 123 && c1 == 0)
        q[32 * b0 + t0] = 0.;
      if (32 * b0 + t0 <= 123)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 115); c3 += 1)
          q[32 * b0 + t0] = (q[32 * b0 + t0] + (A[32 * b0 + t0][c1 + c3] * const_p[c1 + c3]));
      __syncthreads();
    }
}
void init_const_scop_0(double p[116],double r[124])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_p), p, (116) * sizeof(double)));
cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_r), r, (124) * sizeof(double)));


}

