#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "bicg_kernel.hu"
__global__ void kernel0(float A[124][116], float r[124], float s[116])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_r[32];
    __shared__ float shared_s[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      for (int c1 = 0; c1 <= 123; c1 += 32) {
        if (t0 + c1 <= 123)
          shared_r[t0] = r[t0 + c1];
        __syncthreads();
        if (32 * b0 + t0 <= 115 && c1 == 0)
          shared_s[t0] = 0;
        if (32 * b0 + t0 <= 115)
          for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 123); c3 += 1)
            shared_s[t0] = (shared_s[t0] + (shared_r[c3] * A[c1 + c3][32 * b0 + t0]));
        __syncthreads();
      }
      if (32 * b0 + t0 <= 115)
        s[32 * b0 + t0] = shared_s[t0];
    }
}
__global__ void kernel1(float A[124][116], float p[116], float q[124])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_A[32][32];
    __shared__ float shared_p[32];
    __shared__ float shared_q[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      for (int c1 = 0; c1 <= 115; c1 += 32) {
        if (t0 + c1 <= 115) {
          for (int c2 = 0; c2 <= ppcg_min(31, -32 * b0 + 123); c2 += 1)
            shared_A[c2][t0] = A[32 * b0 + c2][t0 + c1];
          shared_p[t0] = p[t0 + c1];
        }
        __syncthreads();
        if (32 * b0 + t0 <= 123 && c1 == 0)
          shared_q[t0] = 0.F;
        if (32 * b0 + t0 <= 123)
          for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 115); c3 += 1)
            shared_q[t0] = (shared_q[t0] + (shared_A[t0][c3] * shared_p[c3]));
        __syncthreads();
      }
      if (32 * b0 + t0 <= 123)
        q[32 * b0 + t0] = shared_q[t0];
    }
}
