#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "bicg_kernel.hu"
__global__ void kernel0(double A[124][116], double r[124], double s[116])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 123; c1 += 32) {
      if (32 * b0 + t0 <= 115 && c1 == 0)
        s[32 * b0 + t0] = 0;
      if (32 * b0 + t0 <= 115)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 123); c3 += 1)
          s[32 * b0 + t0] = (s[32 * b0 + t0] + (r[c1 + c3] * A[c1 + c3][32 * b0 + t0]));
      __syncthreads();
    }
}
__global__ void kernel1(double A[124][116], double p[116], double q[124])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 115; c1 += 32) {
      if (32 * b0 + t0 <= 123 && c1 == 0)
        q[32 * b0 + t0] = 0.;
      if (32 * b0 + t0 <= 123)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 115); c3 += 1)
          q[32 * b0 + t0] = (q[32 * b0 + t0] + (A[32 * b0 + t0][c1 + c3] * p[c1 + c3]));
      __syncthreads();
    }
}
