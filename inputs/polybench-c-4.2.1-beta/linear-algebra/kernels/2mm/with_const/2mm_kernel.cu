#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "2mm_kernel.hu"
__global__ void kernel0(double A[40][70], double alpha, double tmp[40][50])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 69; c2 += 32) {
      if (32 * b0 + t0 <= 39)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 49); c4 += 16) {
          if (c2 == 0)
            tmp[32 * b0 + t0][32 * b1 + c4] = 0.;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 69); c5 += 1)
            tmp[32 * b0 + t0][32 * b1 + c4] += ((alpha * A[32 * b0 + t0][c2 + c5]) * const_B[c2 + c5][32 * b1 + c4]);
        }
      __syncthreads();
    }
}
__global__ void kernel1(double D[40][80], double beta, double tmp[40][50])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 49; c2 += 32) {
      if (32 * b0 + t0 <= 39)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 79); c4 += 16) {
          if (c2 == 0)
            D[32 * b0 + t0][32 * b1 + c4] *= beta;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 49); c5 += 1)
            D[32 * b0 + t0][32 * b1 + c4] += (tmp[32 * b0 + t0][c2 + c5] * const_C[c2 + c5][32 * b1 + c4]);
        }
      __syncthreads();
    }
}
void init_const_scop_0(double B[70][50],double C[50][80])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_B), B, (70) * (50) * sizeof(double)));
cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_C), C, (50) * (80) * sizeof(double)));


}

