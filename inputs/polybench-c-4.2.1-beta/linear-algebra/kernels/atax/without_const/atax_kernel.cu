#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "atax_kernel.hu"
__global__ void kernel0(double A[116][124], double tmp[116], double x[124])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 123; c1 += 32) {
      if (32 * b0 + t0 <= 115 && c1 == 0)
        tmp[32 * b0 + t0] = 0.;
      if (32 * b0 + t0 <= 115)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 123); c3 += 1)
          tmp[32 * b0 + t0] = (tmp[32 * b0 + t0] + (A[32 * b0 + t0][c1 + c3] * x[c1 + c3]));
      __syncthreads();
    }
}
__global__ void kernel1(double A[116][124], double tmp[116], double y[124])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 115; c1 += 32) {
      if (32 * b0 + t0 <= 123 && c1 == 0)
        y[32 * b0 + t0] = 0;
      if (32 * b0 + t0 <= 123)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 115); c3 += 1)
          y[32 * b0 + t0] = (y[32 * b0 + t0] + (A[c1 + c3][32 * b0 + t0] * tmp[c1 + c3]));
      __syncthreads();
    }
}
