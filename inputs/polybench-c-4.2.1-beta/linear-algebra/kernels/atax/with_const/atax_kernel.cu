#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "atax_kernel.hu"
__global__ void kernel0(double A[116][124], double tmp[116])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 123; c1 += 32) {
      if (32 * b0 + t0 <= 115 && c1 == 0)
        tmp[32 * b0 + t0] = 0.;
      if (32 * b0 + t0 <= 115)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 123); c3 += 1)
          tmp[32 * b0 + t0] = (tmp[32 * b0 + t0] + (A[32 * b0 + t0][c1 + c3] * const_x[c1 + c3]));
      __syncthreads();
    }
}
__global__ void kernel1(double A[116][124], double tmp[116], double y[124])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 115; c1 += 32) {
      if (32 * b0 + t0 <= 123 && c1 == 0)
        y[32 * b0 + t0] = 0;
      if (32 * b0 + t0 <= 123)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 115); c3 += 1)
          y[32 * b0 + t0] = (y[32 * b0 + t0] + (A[c1 + c3][32 * b0 + t0] * tmp[c1 + c3]));
      __syncthreads();
    }
}
void init_const_scop_0(double x[124])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_x), x, (124) * sizeof(double)));


}

