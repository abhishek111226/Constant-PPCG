#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "atax_kernel.hu"
__global__ void kernel0(float A[116][124], float tmp[116], float x[124])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_A[32][32];
    __shared__ float shared_tmp[32];
    __shared__ float shared_x[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      for (int c1 = 0; c1 <= 123; c1 += 32) {
        if (t0 + c1 <= 123) {
          for (int c2 = 0; c2 <= ppcg_min(31, -32 * b0 + 115); c2 += 1)
            shared_A[c2][t0] = A[32 * b0 + c2][t0 + c1];
          shared_x[t0] = x[t0 + c1];
        }
        __syncthreads();
        if (32 * b0 + t0 <= 115 && c1 == 0)
          shared_tmp[t0] = 0.F;
        if (32 * b0 + t0 <= 115)
          for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 123); c3 += 1)
            shared_tmp[t0] = (shared_tmp[t0] + (shared_A[t0][c3] * shared_x[c3]));
        __syncthreads();
      }
      if (32 * b0 + t0 <= 115)
        tmp[32 * b0 + t0] = shared_tmp[t0];
    }
}
__global__ void kernel1(float A[116][124], float tmp[116], float y[124])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_tmp[32];
    __shared__ float shared_y[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      for (int c1 = 0; c1 <= 115; c1 += 32) {
        if (t0 + c1 <= 115)
          shared_tmp[t0] = tmp[t0 + c1];
        __syncthreads();
        if (32 * b0 + t0 <= 123 && c1 == 0)
          shared_y[t0] = 0;
        if (32 * b0 + t0 <= 123)
          for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 115); c3 += 1)
            shared_y[t0] = (shared_y[t0] + (A[c1 + c3][32 * b0 + t0] * shared_tmp[c3]));
        __syncthreads();
      }
      if (32 * b0 + t0 <= 123)
        y[32 * b0 + t0] = shared_y[t0];
    }
}
