#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "trmm_kernel.hu"
__global__ void kernel0(double B[60][80])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    for (int c1 = 0; c1 <= 58; c1 += 32)
      for (int c2 = c1; c2 <= 59; c2 += 32) {
        if (32 * b0 + t0 <= 79)
          for (int c4 = 0; c4 <= ppcg_min(ppcg_min(31, -c1 + 58), -c1 + c2 + 30); c4 += 1)
            for (int c5 = ppcg_max(0, c1 - c2 + c4 + 1); c5 <= ppcg_min(31, -c2 + 59); c5 += 1)
              B[c1 + c4][32 * b0 + t0] += (const_A[c2 + c5][c1 + c4] * B[c2 + c5][32 * b0 + t0]);
        __syncthreads();
      }
}
__global__ void kernel1(double B[60][80], double alpha)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    if (32 * b0 + t0 <= 59)
      for (int c3 = t1; c3 <= ppcg_min(31, -32 * b1 + 79); c3 += 16)
        B[32 * b0 + t0][32 * b1 + c3] = (alpha * B[32 * b0 + t0][32 * b1 + c3]);
}
void init_const_scop_0(double A[60][60])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_A), A, (60) * (60) * sizeof(double)));


}

