#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "gemver_kernel.hu"
__global__ void kernel0(double A[120][120], double u1[120], double u2[120], double v1[120], double v2[120])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    if (32 * b0 + t0 <= 119)
      for (int c3 = t1; c3 <= ppcg_min(31, -32 * b1 + 119); c3 += 16)
        A[32 * b0 + t0][32 * b1 + c3] = ((A[32 * b0 + t0][32 * b1 + c3] + (u1[32 * b0 + t0] * v1[32 * b1 + c3])) + (u2[32 * b0 + t0] * v2[32 * b1 + c3]));
}
__global__ void kernel1(double A[120][120], double beta, double x[120], double y[120], double z[120])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 119; c1 += 32) {
      if (32 * b0 + t0 <= 119) {
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 119); c3 += 1)
          x[32 * b0 + t0] = (x[32 * b0 + t0] + ((beta * A[c1 + c3][32 * b0 + t0]) * y[c1 + c3]));
        if (c1 == 96)
          x[32 * b0 + t0] = (x[32 * b0 + t0] + z[32 * b0 + t0]);
      }
      __syncthreads();
    }
}
__global__ void kernel2(double A[120][120], double alpha, double w[120], double x[120])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 119; c1 += 32) {
      if (32 * b0 + t0 <= 119)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 119); c3 += 1)
          w[32 * b0 + t0] = (w[32 * b0 + t0] + ((alpha * A[32 * b0 + t0][c1 + c3]) * x[c1 + c3]));
      __syncthreads();
    }
}
