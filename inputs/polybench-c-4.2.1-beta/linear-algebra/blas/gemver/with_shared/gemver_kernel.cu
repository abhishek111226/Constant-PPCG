#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "gemver_kernel.hu"
__global__ void kernel0(float A[120][120], float u1[120], float u2[120], float v1[120], float v2[120])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_u1[32];
    __shared__ float shared_u2[32];
    __shared__ float shared_v1[32];
    __shared__ float shared_v2[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      if (t0 == 0) {
        for (int c0 = t1; c0 <= ppcg_min(31, -32 * b0 + 119); c0 += 16)
          shared_u1[c0] = u1[32 * b0 + c0];
        for (int c0 = t1; c0 <= ppcg_min(31, -32 * b0 + 119); c0 += 16)
          shared_u2[c0] = u2[32 * b0 + c0];
        for (int c0 = t1; c0 <= ppcg_min(31, -32 * b1 + 119); c0 += 16)
          shared_v1[c0] = v1[32 * b1 + c0];
        for (int c0 = t1; c0 <= ppcg_min(31, -32 * b1 + 119); c0 += 16)
          shared_v2[c0] = v2[32 * b1 + c0];
      }
      __syncthreads();
      if (32 * b0 + t0 <= 119)
        for (int c3 = t1; c3 <= ppcg_min(31, -32 * b1 + 119); c3 += 16)
          A[32 * b0 + t0][32 * b1 + c3] = ((A[32 * b0 + t0][32 * b1 + c3] + (shared_u1[t0] * shared_v1[c3])) + (shared_u2[t0] * shared_v2[c3]));
    }
}
__global__ void kernel1(float A[120][120], float beta, float x[120], float y[120], float z[120])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_x[32];
    __shared__ float shared_y[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      if (32 * b0 + t0 <= 119)
        shared_x[t0] = x[32 * b0 + t0];
      __syncthreads();
      for (int c1 = 0; c1 <= 119; c1 += 32) {
        if (t0 + c1 <= 119)
          shared_y[t0] = y[t0 + c1];
        __syncthreads();
        if (32 * b0 + t0 <= 119) {
          for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 119); c3 += 1)
            shared_x[t0] = (shared_x[t0] + ((beta * A[c1 + c3][32 * b0 + t0]) * shared_y[c3]));
          if (c1 == 96)
            shared_x[t0] = (shared_x[t0] + z[32 * b0 + t0]);
        }
        __syncthreads();
      }
      if (32 * b0 + t0 <= 119)
        x[32 * b0 + t0] = shared_x[t0];
    }
}
__global__ void kernel2(float A[120][120], float alpha, float w[120], float x[120])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_A[32][32];
    __shared__ float shared_w[32];
    __shared__ float shared_x[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      if (32 * b0 + t0 <= 119)
        shared_w[t0] = w[32 * b0 + t0];
      for (int c1 = 0; c1 <= 119; c1 += 32) {
        if (t0 + c1 <= 119) {
          for (int c2 = 0; c2 <= ppcg_min(31, -32 * b0 + 119); c2 += 1)
            shared_A[c2][t0] = A[32 * b0 + c2][t0 + c1];
          shared_x[t0] = x[t0 + c1];
        }
        __syncthreads();
        if (32 * b0 + t0 <= 119)
          for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 119); c3 += 1)
            shared_w[t0] = (shared_w[t0] + ((alpha * shared_A[t0][c3]) * shared_x[c3]));
        __syncthreads();
      }
      if (32 * b0 + t0 <= 119)
        w[32 * b0 + t0] = shared_w[t0];
    }
}
