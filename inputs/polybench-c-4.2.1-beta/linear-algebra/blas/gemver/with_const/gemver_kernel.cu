#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "gemver_kernel.hu"
__global__ void kernel0(double A[120][120], double u1[120], double u2[120])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    if (32 * b0 + t0 <= 119)
      for (int c3 = t1; c3 <= ppcg_min(31, -32 * b1 + 119); c3 += 16)
        A[32 * b0 + t0][32 * b1 + c3] = ((A[32 * b0 + t0][32 * b1 + c3] + (u1[32 * b0 + t0] * const_v1[32 * b1 + c3])) + (u2[32 * b0 + t0] * const_v2[32 * b1 + c3]));
}
__global__ void kernel1(double A[120][120], double beta, double x[120], double z[120])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 119; c1 += 32) {
      if (32 * b0 + t0 <= 119) {
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 119); c3 += 1)
          x[32 * b0 + t0] = (x[32 * b0 + t0] + ((beta * A[c1 + c3][32 * b0 + t0]) * const_y[c1 + c3]));
        if (c1 == 96)
          x[32 * b0 + t0] = (x[32 * b0 + t0] + z[32 * b0 + t0]);
      }
      __syncthreads();
    }
}
__global__ void kernel2(double A[120][120], double alpha, double w[120], double x[120])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 119; c1 += 32) {
      if (32 * b0 + t0 <= 119)
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 119); c3 += 1)
          w[32 * b0 + t0] = (w[32 * b0 + t0] + ((alpha * A[32 * b0 + t0][c1 + c3]) * x[c1 + c3]));
      __syncthreads();
    }
}
void init_const_scop_0(double v1[120],double v2[120],double y[120])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_v1), v1, (120) * sizeof(double)));
cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_v2), v2, (120) * sizeof(double)));
cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_y), y, (120) * sizeof(double)));


}

