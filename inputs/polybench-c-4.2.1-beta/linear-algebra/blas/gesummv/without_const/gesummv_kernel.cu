#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "gesummv_kernel.hu"
__global__ void kernel0(double A[90][90], double B[90][90], double alpha, double beta, double tmp[90], double x[90], double y[90])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 89; c1 += 32) {
      if (32 * b0 + t0 <= 89 && c1 == 0)
        y[32 * b0 + t0] = 0.;
      if (32 * b0 + t0 <= 89) {
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 89); c3 += 1) {
          y[32 * b0 + t0] = ((B[32 * b0 + t0][c1 + c3] * x[c1 + c3]) + y[32 * b0 + t0]);
          if (c1 == 0 && c3 == 0)
            tmp[32 * b0 + t0] = 0.;
          tmp[32 * b0 + t0] = ((A[32 * b0 + t0][c1 + c3] * x[c1 + c3]) + tmp[32 * b0 + t0]);
        }
        if (c1 == 64)
          y[32 * b0 + t0] = ((alpha * tmp[32 * b0 + t0]) + (beta * y[32 * b0 + t0]));
      }
      __syncthreads();
    }
}
