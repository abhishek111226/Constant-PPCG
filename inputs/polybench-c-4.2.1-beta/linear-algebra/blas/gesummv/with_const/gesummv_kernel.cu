#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "gesummv_kernel.hu"
__global__ void kernel0(double A[90][90], double B[90][90], double alpha, double beta, double tmp[90], double y[90])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 89; c1 += 32) {
      if (32 * b0 + t0 <= 89 && c1 == 0)
        y[32 * b0 + t0] = 0.;
      if (32 * b0 + t0 <= 89) {
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 89); c3 += 1) {
          y[32 * b0 + t0] = ((B[32 * b0 + t0][c1 + c3] * const_x[c1 + c3]) + y[32 * b0 + t0]);
          if (c1 == 0 && c3 == 0)
            tmp[32 * b0 + t0] = 0.;
          tmp[32 * b0 + t0] = ((A[32 * b0 + t0][c1 + c3] * const_x[c1 + c3]) + tmp[32 * b0 + t0]);
        }
        if (c1 == 64)
          y[32 * b0 + t0] = ((alpha * tmp[32 * b0 + t0]) + (beta * y[32 * b0 + t0]));
      }
      __syncthreads();
    }
}
void init_const_scop_0(double x[90])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_x), x, (90) * sizeof(double)));


}

