#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "gesummv_kernel.hu"
__global__ void kernel0(float A[90][90], float B[90][90], float alpha, float beta, float tmp[90], float x[90], float y[90])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_A[32][32];
    __shared__ float shared_B[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 0; c1 <= 89; c1 += 32) {
      if (t0 + c1 <= 89) {
        for (int c2 = 0; c2 <= ppcg_min(31, -32 * b0 + 89); c2 += 1)
          shared_A[c2][t0] = A[32 * b0 + c2][t0 + c1];
        for (int c2 = 0; c2 <= ppcg_min(31, -32 * b0 + 89); c2 += 1)
          shared_B[c2][t0] = B[32 * b0 + c2][t0 + c1];
      }
      __syncthreads();
      if (32 * b0 + t0 <= 89 && c1 == 0)
        y[32 * b0 + t0] = 0.F;
      if (32 * b0 + t0 <= 89) {
        for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 89); c3 += 1) {
          y[32 * b0 + t0] = ((shared_B[t0][c3] * x[c1 + c3]) + y[32 * b0 + t0]);
          if (c1 == 0 && c3 == 0)
            tmp[32 * b0 + t0] = 0.F;
          tmp[32 * b0 + t0] = ((shared_A[t0][c3] * x[c1 + c3]) + tmp[32 * b0 + t0]);
        }
        if (c1 == 64)
          y[32 * b0 + t0] = ((alpha * tmp[32 * b0 + t0]) + (beta * y[32 * b0 + t0]));
      }
      __syncthreads();
    }
}
