#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "gemm_kernel.hu"
__global__ void kernel0(float A[60][80], float B[80][70], float C[60][70], float alpha, float beta)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_A[32][32];
    __shared__ float shared_B[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 79; c2 += 32) {
      if (32 * b0 + t0 <= 59)
        for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 79); c4 += 16)
          shared_A[t0][c4] = A[32 * b0 + t0][c2 + c4];
      if (t0 + c2 <= 79)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 69); c4 += 16)
          shared_B[t0][c4] = B[t0 + c2][32 * b1 + c4];
      __syncthreads();
      if (32 * b0 + t0 <= 59)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 69); c4 += 16) {
          if (c2 == 0)
            C[32 * b0 + t0][32 * b1 + c4] *= beta;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 79); c5 += 1)
            C[32 * b0 + t0][32 * b1 + c4] += ((alpha * shared_A[t0][c5]) * shared_B[c5][c4]);
        }
      __syncthreads();
    }
}
