#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "gemm_kernel.hu"
__global__ void kernel0(double A[60][80], double C[60][70], double alpha, double beta)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 79; c2 += 32) {
      if (32 * b0 + t0 <= 59)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 69); c4 += 16) {
          if (c2 == 0)
            C[32 * b0 + t0][32 * b1 + c4] *= beta;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 79); c5 += 1)
            C[32 * b0 + t0][32 * b1 + c4] += ((alpha * A[32 * b0 + t0][c2 + c5]) * const_B[c2 + c5][32 * b1 + c4]);
        }
      __syncthreads();
    }
}
void init_const_scop_0(double B[80][70])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_B), B, (80) * (70) * sizeof(double)));


}

