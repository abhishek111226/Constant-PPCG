#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "durbin_kernel.hu"
__global__ void kernel0(double r[120], double *sum, double y[120], int c0)
{
    double private_sum;

    {
      private_sum = *sum;
      for (int c1 = 0; c1 < (c0 - 117) / 472; c1 += 1)
        private_sum += (r[((c0 - 589) / 472) - c1] * y[c1]);
      *sum = private_sum;
    }
}
__global__ void kernel1(double *alpha, double *beta, int c0)
{

    beta[0] = ((1 - (alpha[0] * alpha[0])) * beta[0]);
}
__global__ void kernel2(double *alpha, double y[120], int c0)
{

    y[(c0 - 472) / 471] = alpha[0];
}
__global__ void kernel3(double *alpha, double *beta, double r[120], double *sum, int c0)
{

    alpha[0] = ((-(r[(c0 - 235) / 472] + sum[0])) / beta[0]);
}
__global__ void kernel4(double *sum, int c0)
{

    sum[0] = 0.;
}
__global__ void kernel5(double y[120], double z[119], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (c0 >= 15104 * b0 + 472 * t0 + 943)
      y[32 * b0 + t0] = z[32 * b0 + t0];
}
__global__ void kernel6(double *alpha, double y[120], double z[119], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (c0 >= 15104 * b0 + 472 * t0 + 825)
      z[32 * b0 + t0] = (y[32 * b0 + t0] + (alpha[0] * y[((c0 - 825) / 472) - 32 * b0 - t0]));
}
