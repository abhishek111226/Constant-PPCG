#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "durbin_kernel.hu"
__global__ void kernel0(double *sum, double y[120], int c0)
{
    double private_sum;

    {
      private_sum = *sum;
      for (int c1 = 0; c1 < (c0 - 117) / 472; c1 += 1)
        private_sum += (const_r[((c0 - 589) / 472) - c1] * y[c1]);
      *sum = private_sum;
    }
}
__global__ void kernel1(double *alpha, double *beta, int c0)
{

    beta[0] = ((1 - (alpha[0] * alpha[0])) * beta[0]);
}
__global__ void kernel2(double *alpha, double y[120], int c0)
{

    y[(c0 - 472) / 471] = alpha[0];
}
__global__ void kernel3(double *alpha, double *beta, double *sum, int c0)
{

    alpha[0] = ((-(const_r[(c0 - 235) / 472] + sum[0])) / beta[0]);
}
__global__ void kernel4(double *sum, int c0)
{

    sum[0] = 0.;
}
__global__ void kernel5(double y[120], double z[119], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (c0 >= 15104 * b0 + 472 * t0 + 943)
      y[32 * b0 + t0] = z[32 * b0 + t0];
}
__global__ void kernel6(double *alpha, double y[120], double z[119], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (c0 >= 15104 * b0 + 472 * t0 + 825)
      z[32 * b0 + t0] = (y[32 * b0 + t0] + (alpha[0] * y[((c0 - 825) / 472) - 32 * b0 - t0]));
}
void init_const_scop_0(double r[120])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_r), r, (120) * sizeof(double)));


}

