#include <stdio.h>
#define HOSTCODE true 
#include "durbin_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* durbin.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "durbin.h"


/* Array initialization. */
static
void init_array (int n,
		 DATA_TYPE POLYBENCH_1D(r,N,n))
{
  int i, j;

  for (i = 0; i < n; i++)
    {
      r[i] = (n+1-i);
    }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(y,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("y");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, y[i]);
  }
  POLYBENCH_DUMP_END("y");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_durbin(int n,
		   DATA_TYPE POLYBENCH_1D(r,N,n),
		   DATA_TYPE POLYBENCH_1D(y,N,n))
{
 DATA_TYPE z[N];
 DATA_TYPE alpha;
 DATA_TYPE beta;
 DATA_TYPE sum;

 int i,k;

 {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

   float *dev_alpha;
   float *dev_beta;
   float *dev_r;
   float *dev_sum;
   float *dev_y;
   float *dev_z;
   
   cudaCheckReturn(hipMalloc((void **) &dev_alpha, sizeof(float)));
   cudaCheckReturn(hipMalloc((void **) &dev_beta, sizeof(float)));
   cudaCheckReturn(hipMalloc((void **) &dev_r, (120) * sizeof(float)));
   cudaCheckReturn(hipMalloc((void **) &dev_sum, sizeof(float)));
   cudaCheckReturn(hipMalloc((void **) &dev_y, (120) * sizeof(float)));
   cudaCheckReturn(hipMalloc((void **) &dev_z, (119) * sizeof(float)));
   
   
   beta = 1.F;
   alpha = (-r[0]);
   y[0] = (-r[0]);
   cudaCheckReturn(hipMemcpy(dev_alpha, &alpha, sizeof(float), hipMemcpyHostToDevice));
   cudaCheckReturn(hipMemcpy(dev_beta, &beta, sizeof(float), hipMemcpyHostToDevice));
   cudaCheckReturn(hipMemcpy(dev_r, r, (120) * sizeof(float), hipMemcpyHostToDevice));
   cudaCheckReturn(hipMemcpy(dev_y, y, (120) * sizeof(float), hipMemcpyHostToDevice));
   for (int c0 = 471; c0 <= 56639; c0 += 1) {
     if ((c0 - 117) % 472 == 0)
       {
         dim3 k0_dimBlock;
         dim3 k0_dimGrid;
         kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_r, dev_sum, dev_y, c0);
         cudaCheckKernel();
       }
       
       
     if (c0 <= 56049 && (c0 - 119) % 470 == 0)
       {
         dim3 k1_dimBlock;
         dim3 k1_dimGrid;
         kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_alpha, dev_beta, c0);
         cudaCheckKernel();
       }
       
       
     if (c0 >= 943 && (c0 - 1) % 471 == 0) {
       {
         dim3 k2_dimBlock;
         dim3 k2_dimGrid;
         kernel2 <<<k2_dimGrid, k2_dimBlock>>> (dev_alpha, dev_y, c0);
         cudaCheckKernel();
       }
       
       
     } else if ((c0 - 235) % 472 == 0) {
       {
         dim3 k3_dimBlock;
         dim3 k3_dimGrid;
         kernel3 <<<k3_dimGrid, k3_dimBlock>>> (dev_alpha, dev_beta, dev_r, dev_sum, c0);
         cudaCheckKernel();
       }
       
       
     } else if (c0 <= 56049 && c0 % 471 == 0)
       {
         dim3 k4_dimBlock;
         dim3 k4_dimGrid;
         kernel4 <<<k4_dimGrid, k4_dimBlock>>> (dev_sum, c0);
         cudaCheckKernel();
       }
       
       
     if (c0 >= 943 && (c0 - 471) % 472 == 0) {
       {
         dim3 k5_dimBlock(32);
         dim3 k5_dimGrid(4);
         kernel5 <<<k5_dimGrid, k5_dimBlock>>> (dev_y, dev_z, c0);
         cudaCheckKernel();
       }
       
       
     } else if ((c0 - 353) % 472 == 0)
       {
         dim3 k6_dimBlock(32);
         dim3 k6_dimGrid(4);
         kernel6 <<<k6_dimGrid, k6_dimBlock>>> (dev_alpha, dev_y, dev_z, c0);
         cudaCheckKernel();
       }
       
       
   }
   cudaCheckReturn(hipMemcpy(y, dev_y, (120) * sizeof(float), hipMemcpyDeviceToHost));
   
   
   cudaCheckReturn(hipFree(dev_alpha));
   cudaCheckReturn(hipFree(dev_beta));
   cudaCheckReturn(hipFree(dev_r));
   cudaCheckReturn(hipFree(dev_sum));
   cudaCheckReturn(hipFree(dev_y));
   cudaCheckReturn(hipFree(dev_z));
 }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  POLYBENCH_1D_ARRAY_DECL(r, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y, DATA_TYPE, N, n);


  /* Initialize array(s). */
  init_array (n, POLYBENCH_ARRAY(r));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_durbin (n,
		 POLYBENCH_ARRAY(r),
		 POLYBENCH_ARRAY(y));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(y)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(r);
  POLYBENCH_FREE_ARRAY(y);

  return 0;
}
