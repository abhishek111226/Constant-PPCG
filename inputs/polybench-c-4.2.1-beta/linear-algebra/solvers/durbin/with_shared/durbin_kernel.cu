#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "durbin_kernel.hu"
__global__ void kernel0(float r[120], float *sum, float y[120], int c0)
{
    float private_sum;

    {
      private_sum = *sum;
      for (int c1 = 0; c1 < (c0 - 117) / 472; c1 += 1)
        private_sum += (r[((c0 - 589) / 472) - c1] * y[c1]);
      *sum = private_sum;
    }
}
__global__ void kernel1(float *alpha, float *beta, int c0)
{

    beta[0] = ((1 - (alpha[0] * alpha[0])) * beta[0]);
}
__global__ void kernel2(float *alpha, float y[120], int c0)
{

    y[(c0 - 472) / 471] = alpha[0];
}
__global__ void kernel3(float *alpha, float *beta, float r[120], float *sum, int c0)
{

    alpha[0] = ((-(r[(c0 - 235) / 472] + sum[0])) / beta[0]);
}
__global__ void kernel4(float *sum, int c0)
{

    sum[0] = 0.F;
}
__global__ void kernel5(float y[120], float z[119], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (c0 >= 15104 * b0 + 472 * t0 + 943)
      y[32 * b0 + t0] = z[32 * b0 + t0];
}
__global__ void kernel6(float *alpha, float y[120], float z[119], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_alpha;
    __shared__ float shared_y_1[32];

    {
      if (t0 == 0 && c0 >= 15104 * b0 + 825)
        shared_alpha = *alpha;
      if (472 * t0 + c0 >= 15104 * b0 + 15457)
        shared_y_1[t0] = y[((c0 - 15457) / 472) - 32 * b0 + t0];
      __syncthreads();
      if (c0 >= 15104 * b0 + 472 * t0 + 825)
        z[32 * b0 + t0] = (y[32 * b0 + t0] + (shared_alpha * shared_y_1[-t0 + 31]));
    }
}
