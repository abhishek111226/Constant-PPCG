#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "ludcmp_kernel.hu"
__global__ void kernel0(float A[120][120], int c0)
{
    float private_w;

    for (int c1 = 0; c1 < c0; c1 += 1) {
      private_w = A[c0][c1];
      for (int c2 = 0; c2 < c1; c2 += 1)
        private_w -= (A[c0][c2] * A[c2][c1]);
      A[c0][c1] = (private_w / A[c1][c1]);
    }
}
__global__ void kernel1(float A[120][120], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_A_0[1][32];
    __shared__ float shared_A_1[1][119];
    float private_w;

    {
      if (32 * b0 + t0 <= 119)
        shared_A_0[0][t0] = A[c0][32 * b0 + t0];
      for (int c2 = t0; c2 <= 118; c2 += 32)
        shared_A_1[0][c2] = A[c0][c2];
      __syncthreads();
      if (32 * b0 + t0 <= 119 && 32 * b0 + t0 >= c0) {
        private_w = shared_A_0[0][t0];
        for (int c3 = 0; c3 < c0; c3 += 1)
          private_w -= (shared_A_1[0][c3] * A[c3][32 * b0 + t0]);
        shared_A_0[0][t0] = private_w;
      }
      __syncthreads();
      if (32 * b0 + t0 <= 119 && 32 * b0 + t0 >= c0)
        A[c0][32 * b0 + t0] = shared_A_0[0][t0];
    }
}
__global__ void kernel2(float A[120][120], float b[120], float y[120])
{
    float private_w;

    for (int c0 = 0; c0 <= 119; c0 += 1) {
      private_w = b[c0];
      for (int c1 = 0; c1 < c0; c1 += 1)
        private_w -= (A[c0][c1] * y[c1]);
      y[c0] = private_w;
    }
}
__global__ void kernel3(float A[120][120], float x[120], float y[120])
{
    float private_w;

    for (int c0 = -119; c0 <= 0; c0 += 1) {
      private_w = y[-c0];
      for (int c1 = -c0 + 1; c1 <= 119; c1 += 1)
        private_w -= (A[-c0][c1] * x[c1]);
      x[-c0] = (private_w / A[-c0][-c0]);
    }
}
