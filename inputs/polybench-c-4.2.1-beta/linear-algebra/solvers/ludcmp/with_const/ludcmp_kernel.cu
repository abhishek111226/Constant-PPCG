#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "ludcmp_kernel.hu"
__global__ void kernel0(double A[120][120], int c0)
{
    double private_w;

    for (int c1 = 0; c1 < c0; c1 += 1) {
      private_w = A[c0][c1];
      for (int c2 = 0; c2 < c1; c2 += 1)
        private_w -= (A[c0][c2] * A[c2][c1]);
      A[c0][c1] = (private_w / A[c1][c1]);
    }
}
__global__ void kernel1(double A[120][120], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    double private_w;

    if (32 * b0 + t0 <= 119 && 32 * b0 + t0 >= c0) {
      private_w = A[c0][32 * b0 + t0];
      for (int c3 = 0; c3 < c0; c3 += 1)
        private_w -= (A[c0][c3] * A[c3][32 * b0 + t0]);
      A[c0][32 * b0 + t0] = private_w;
    }
}
__global__ void kernel2(double A[120][120], double y[120])
{
    double private_w;

    for (int c0 = 0; c0 <= 119; c0 += 1) {
      private_w = const_b[c0];
      for (int c1 = 0; c1 < c0; c1 += 1)
        private_w -= (A[c0][c1] * y[c1]);
      y[c0] = private_w;
    }
}
__global__ void kernel3(double A[120][120], double x[120], double y[120])
{
    double private_w;

    for (int c0 = -119; c0 <= 0; c0 += 1) {
      private_w = y[-c0];
      for (int c1 = -c0 + 1; c1 <= 119; c1 += 1)
        private_w -= (A[-c0][c1] * x[c1]);
      x[-c0] = (private_w / A[-c0][-c0]);
    }
}
void init_const_scop_0(double b[120])
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


cudaCheckReturn(hipMemcpyToSymbol(HIP_SYMBOL(const_b), b, (120) * sizeof(double)));


}

