#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "mvt_kernel.hu"
#define _PB_N 100
int x1[100];
int x2[100];
int y_1[100];
int y_2[100];
int A[100][100];
int n = 100;
void init_array()
{
  int i, j;

  for (i = 0; i < n; i++)
    {
      x1[i] = (int) (i % n) ;
      x2[i] = (int) ((i + 1) % n) ;
      y_1[i] = (int) ((i + 3) % n) ;
      y_2[i] = (int) ((i + 4) % n);
      for (j = 0; j < n; j++)
	A[i][j] = (int) (i*j % n);
    }
}

int main()
{
  int i,j;
  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    int *dev_A;
    int *dev_x1;
    int *dev_x2;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (100) * (100) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_x1, (100) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_x2, (100) * sizeof(int)));
    
    
    init_const_scop_0(y_1,y_2);
    
    cudaCheckReturn(hipMemcpy(dev_A, A, (100) * (100) * sizeof(int), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x1, x1, (100) * sizeof(int), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x2, x2, (100) * sizeof(int), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(32);
      dim3 k0_dimGrid(4);
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_x1);
      cudaCheckKernel();
    }
    
    
    {
      dim3 k1_dimBlock(32);
      dim3 k1_dimGrid(4);
      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, dev_x2);
      cudaCheckKernel();
    }
    
    
    cudaCheckReturn(hipMemcpy(x1, dev_x1, (100) * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(x2, dev_x2, (100) * sizeof(int), hipMemcpyDeviceToHost));
    
    
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_x1));
    cudaCheckReturn(hipFree(dev_x2));
  }
}

/*enum RWbar 
{
	write,	0
	read,	1
	invalid,2 
	error,  3 
	none,   4
	read_inside_loop 5
}; */
